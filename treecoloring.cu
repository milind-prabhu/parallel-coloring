
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <algorithm>
#include <string>

using namespace std;

__host__
void writeEdgesToFile(const string& filename, int numNodes) {
	ofstream outFile(filename);

	if (outFile.is_open()) {
		// Write the number of nodes to the file
		outFile << numNodes << endl;

		// Write edges to the file
		for (int i = 2; i <= numNodes; ++i) {
			int parentID = rand() % (i - 1) + 1; // Randomly select a parent ID from existing nodes
			outFile << parentID << " " << i << endl;
		}

		outFile.close();
		// cout << "Edges written to " << filename << endl;
	} else {
		cerr << "Unable to open file: " << filename << endl;
	}
}

__host__
void parseTree(const string& inputFile, int &n, int* &vertices, int* &parents, int* &children, int* &degree)
{
	fstream f;
	f.open(inputFile, ios::in);

	f >> n;

	vertices = new int[n+1];
	parents = new int[n+1];
	degree = new int[n+1];
	children = new int[(n-1)];
	vertices[0] = -1;
	parents[0] = -1;
	parents[1]=-1;
	vector<int> *g = new vector<int>[n + 1];

	int u,v;
	for(int i = 0; i < n-1; i ++)
	{
		f >> u >> v;
		g[u].push_back(v);
		parents[v] = u;
	}

	int ctr = 0;
	for(int i = 1; i <=n; i ++)
	{
		vertices[i] = ctr;
		for(auto &x: g[i])
		{
			children[ctr++] = x;
		}
		degree[i] = (int)g[i].size();
		if(degree[i]==0)
			vertices[i]=-1;
	}
}

__global__
void sixColoringTrees(int n, int* vertices, int* parents, int* colors, int* newColors)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id==0) // Do nothing on id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;

	if(id==1) // No need to change color of root
		return;

	int myColor = colors[id];

	int bitIndex = __ffs(myColor ^ colors[parents[id]]);
	int myBit = (myColor & (1 << (bitIndex-1))) >> (bitIndex-1);
	newColors[id] = ((bitIndex << 1) ^ myBit);
}

__global__
void updateColors(int n, int* colors, int* newColors)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id==0) // Do nothing when id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;
	colors[id] = newColors[id];
	// printf("blahhhhhhhhhhhhhhhhhhhhhhhhhhhhhhhhhhh id:%d, NewColor: %d\n", id, newColors[id]);
}

__global__
void getParentColor(int n, int* parents, int* colors, int* newColors)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id==0) // Do nothing on id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;
	if(id==1)
	{
		newColors[id] = 2; // Change color of root
		return;
	}

	newColors[id] = colors[parents[id]]; // Each node updates its color to the old color of its parents
	// printf("id: %d parent color: %d\n", id, colors[parents[id]]);
}

__global__
void badVertices(int n, int* vertices, int* parents, int* children, int* degree, int* colors, int* newColors, bool* badFlag)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id==0) // Do nothing when id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;
	if(id==1)
	{
		badFlag[id] = 0;
		return;
	}

	int myColor = colors[id];
	if(myColor<=3)
	{
		badFlag[id] = 0;
		return;
	}

	int parentColor = colors[parents[id]];
	if(parentColor <= 3)
	{
		badFlag[id] = 1;
		return;
	}

	int childColor = 0;
	int no_of_children = degree[id];
	int start = vertices[id];
	for(int i=0;i<no_of_children;++i)
	{
		childColor = colors[children[start+i]];
		if(childColor <= 3)
		{
			badFlag[id] = 1;
			return;
		}
	}


	newColors[id] = newColors[id] - 3; // Making its color good, if all the neighbors are bad
	badFlag = 0;
	return;
}

// __global__
// void coloringGoodVertices(int n, int* colors, int* newColors, bool* badFlag)
// {
// 	int id = threadIdx.x + blockIdx.x * blockDim.x;
// 	if(id==0) // Do nothing when id = 0
// 		return;
// 	if(id>n) // Do nothing when id is more than n
// 		return;
// 	if(id==1)
// 		return;

// 	if(colors[id] > 3 && !badFlag[id])
// 	{
// 		newColors[id] = colors[id]-3;
// 		return;
// 	}
// 	if(badFlag[id])
// 		newColors[id] = 0;
// 	return;
// }

// Partially colors bad vertices whose parents are not bad.
__global__
void partialColoringBadVertices(int n, int* vertices, int* parents, int* children, int* degree, int* colors, int* newColors, bool* badFlag)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id==0) // Do nothing when id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;
	if(id==1)
		return;

	if(badFlag[id])
	{
		int parentID = parents[id];
		if(!badFlag[parentID]) // Parent is not a bad vertex
		{
			// badFlag[id] = 0;
			int parentColor = colors[parentID];
			int childColor = 0;
			int no_of_children = degree[id];
			int start = vertices[id];
			for(int i=0;i<no_of_children;++i)
			{
				childColor = colors[children[start+i]];
				if(childColor>0 && childColor<=3)
					break;
			}

			if(parentColor!=1 && childColor!=1)
			{
				newColors[id] = 1;
				return;
			}
			if(parentColor!=2 && childColor!=2)
			{
				newColors[id] = 2;
				return;
			}
			newColors[id] = 3;
			return;
		}
		return;
	}
	return;
}

// Colors all the remaining bad vertices
__global__
void threeColoring(int n, int* vertices, int* parents, int* children, int* degree, int* colors, int* newColors, bool* badFlag)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id==0) // Do nothing when id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;
	if(id==1)
		return;

	if(badFlag[id])
	{
		badFlag[id] = 0;
		int parentID = parents[id];
		int parentColor = colors[parentID];
		int childColor = 0;
		int no_of_children = degree[id];
		int start = vertices[id];
		for(int i=0;i<no_of_children;++i)
		{
			childColor = colors[children[start+i]];
			if(childColor>0 && childColor<=3)
				break;
		}

		if(parentColor!=1 && childColor!=1)
		{
			newColors[id] = 1;
			return;
		}
		if(parentColor!=2 && childColor!=2)
		{
			newColors[id] = 2;
			return;
		}
		newColors[id] = 3;
		return;
	}
}

__global__
void isValidColor(int n, int *vertices, int *parents, int *children, int *degree, int *colors, bool *validColor)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id==0) // Do nothing when id = 0
		return;
	if(id>n) // Do nothing when id is more than n
		return;

	int myColor = colors[id];
	
	int no_of_children=degree[id];
	int start = vertices[id];
	int childColor = 0;
	for(int i=0;i<no_of_children;++i)
	{
		childColor = colors[children[start+i]];
		if(myColor == childColor)
		{
			validColor[0] = false;
			return;
		}
	}

	if(id>1)
	{
		int parentColor = colors[parents[id]];
		if(myColor == parentColor)
		{
			validColor[0] = false;
			return;
		}

	}
}

int main(int argc, char **argv)
{
	srand(static_cast<unsigned>(time(nullptr)));

	string num = argv[1];
	int numNodes = stoi(argv[1], NULL, 10);

	if (numNodes <= 0) {
		cout << "Invalid number of nodes. Please enter a positive integer." << endl;
		return 1;
	}

	// Output the edges directly to a file
	string filename = "random_tree_edges_"+num+".txt";
	writeEdgesToFile(filename, numNodes);

	int n;
	int* vertices = NULL;
	int* parents = NULL;
	int* children = NULL;
	int* degree = NULL;
	parseTree(filename, n, vertices, parents, children, degree);

	int* colors = (int*)malloc((n+1)*sizeof(int));
	bool* badFlag = (bool*)malloc((n+1)*sizeof(bool));
	bool validColor = true;

	// Initialization of colors
	for(int i=1;i<=n;++i)
	{
		colors[i] = i;
		badFlag[i] = 0;
		// newColors[i] = i;
	}

	int L = ceil(log2(n+1));

	// Cuda Memory Allocation
	int *vertices_gpu, *parents_gpu, *children_gpu, *degree_gpu, *colors_gpu, *newColors_gpu;
	bool *badFlag_gpu;
	bool *validColor_gpu;
	// int* L_gpu, delta_gpu;
	if(hipMalloc(&vertices_gpu, (n+1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for vertices" << endl;
	if(hipMalloc(&parents_gpu, (n+1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for parents" << endl;
	if(hipMalloc(&children_gpu, (n-1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for vertices" << endl;
	if(hipMalloc(&degree_gpu, (n+1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for degree" << endl;
	if(hipMalloc(&colors_gpu, (n+1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for colors" << endl;
	if(hipMalloc(&newColors_gpu, (n+1)*sizeof(int)) != hipSuccess)
		cout << "Cannot allocate memory for colors" << endl;
	if(hipMalloc(&badFlag_gpu, (n+1)*sizeof(bool)) != hipSuccess)
		cout << "Cannot allocate memory for bad flags" << endl;
	if(hipMalloc(&validColor_gpu, sizeof(bool)) != hipSuccess)
		cout << "Cannot allocate memory for validColor flag" << endl;

	// cout << "CUDA Memory allocated" << endl;

	// Send data from host to device.
	if(hipMemcpy(vertices_gpu, vertices, (n+1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy vertices to device" << endl;
	if(hipMemcpy(parents_gpu, parents, (n+1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy parents to device" << endl;
	if(hipMemcpy(children_gpu, children, (n-1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy children to device" << endl;
	if(hipMemcpy(degree_gpu, degree, (n+1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy degree to device" << endl;
	if(hipMemcpy(colors_gpu, colors, (n+1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy colors to device" << endl;
	if(hipMemcpy(newColors_gpu, colors, (n+1)*sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy colors to device" << endl;
	if(hipMemcpy(badFlag_gpu, badFlag, (n+1)*sizeof(bool), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy colors to device" << endl;
	if(hipMemcpy(validColor_gpu, &validColor, sizeof(bool), hipMemcpyHostToDevice) != hipSuccess)
		cout << "Cannot copy colors to device" << endl;

	// cout << "Data sent to device" << endl;

	// Set up CUDA event timers
	float elapsedTime = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Mark the beginning of the timed segment
	hipEventRecord(start);

	int blockDim = 256;
	int gridDim = ceil(n/(256.0));

	int count = 0;
	while(L > (ceil(log2(L))+1))
	{
		// cout << "L: " << L << endl;
		sixColoringTrees<<<gridDim, blockDim>>>(n, vertices_gpu, parents_gpu, colors_gpu, newColors_gpu);
		updateColors<<<gridDim, blockDim>>>(n, colors_gpu, newColors_gpu);

		L = (ceil(log2(L))+1);
		count++;
	}

	// cout << "Six coloring done" << endl;

	getParentColor<<<gridDim, blockDim>>>(n, parents_gpu, colors_gpu, newColors_gpu);
	updateColors<<<gridDim, blockDim>>>(n, colors_gpu, newColors_gpu);

	// cout << "Updated color of node to color of parent node" << endl;

	badVertices<<<gridDim, blockDim>>>(n, vertices_gpu, parents_gpu, children_gpu, degree_gpu, colors_gpu, newColors_gpu, badFlag_gpu);
	updateColors<<<gridDim, blockDim>>>(n, colors_gpu, newColors_gpu);

	// cout << "Colored good vertices" << endl;

	partialColoringBadVertices<<<gridDim, blockDim>>>(n, vertices_gpu, parents_gpu, children_gpu, degree_gpu, colors_gpu, newColors_gpu, badFlag_gpu);
	updateColors<<<gridDim, blockDim>>>(n, colors_gpu, newColors_gpu);

	// cout << "partially colored bad vertices" << endl;

	threeColoring<<<gridDim, blockDim>>>(n, vertices_gpu, parents_gpu, children_gpu, degree_gpu, colors_gpu, newColors_gpu, badFlag_gpu);
	updateColors<<<gridDim, blockDim>>>(n, colors_gpu, newColors_gpu);


	// Verification
	isValidColor<<<gridDim, blockDim>>>(n, vertices_gpu, parents_gpu, children_gpu, degree_gpu, colors_gpu, validColor_gpu);


	// Send data from device to host.
	if(hipMemcpy(&validColor, validColor_gpu, sizeof(bool), hipMemcpyDeviceToHost) != hipSuccess)
		cout << "Cannot copy colors from device to host" << endl;

	// Mark the end of the timed segment
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration between the start and stop markers (in milliseconds)
	hipEventElapsedTime(&elapsedTime, start, stop);
	// Clean up event resources
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// cout << "Data copied from device to host" << endl;

	cout << "Time elapsed: " << elapsedTime << " ms" << '\n';
	cout << "Number of iterations: " << count << endl;
	cout << "isValidColoring: " << validColor << endl;

	// for(int i=1;i<=n;++i)
	// {
	// 	cout << "Color of " << i << ": " << colors[i] << endl;
	// }


	hipFree(vertices_gpu);
	hipFree(parents_gpu);
	hipFree(children_gpu);
	hipFree(degree_gpu);
	hipFree(colors_gpu);
	hipFree(newColors_gpu);
	hipFree(badFlag_gpu);
	hipFree(validColor_gpu);
	return 0;
}